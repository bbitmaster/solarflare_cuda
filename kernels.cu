#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

__global__ void do_fire(unsigned int *pData,hiprandState *state,float *firesrc,float *firedest);
__global__ void setup_kernel(hiprandState *state);
hiprandState *devStates;

float *firebuff[2];
int flipstate;

int init_cuda(){

	hipMalloc((void **)&firebuff[0], 600 * 800 *
	                  sizeof(float));

	hipMalloc((void **)&firebuff[1], 600 * 800 *
	                  sizeof(float));

	hipMalloc((void **)&devStates, 600 * 800 *
	                  sizeof(hiprandState));
	setup_kernel<<<600,800>>>(devStates);

	flipstate=0;
	return 0;
}

int run_fire(unsigned int *pData){
	if(flipstate){
		do_fire<<<600,800>>>(pData,devStates,firebuff[0],firebuff[1]);
		flipstate = 0;
	} else {
		do_fire<<<600,800>>>(pData,devStates,firebuff[1],firebuff[0]);
		flipstate = 1;
	}
	return 0;
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void do_fire(unsigned int *pData,hiprandState *state,float *firesrc,float *firedest){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_y = blockIdx.x;
	int thread_x = threadIdx.x;
	int maxwidth = blockDim.x;

	hiprandState localState = state[idx];
	//pData[idx] = ((blockIdx.x&0xff) << 8) + threadIdx.x;

	int rand = hiprand_uniform(&localState)*256;
	rand &= 0xFF;
	state[idx] = localState;

	if(thread_y >= 600-1)
		firedest[idx] = rand;
	__syncthreads();


	if(thread_y < 600-1){
		float avg[4];
		if((thread_x-1) >= 0)avg[1] = firesrc[(thread_y+1)*maxwidth + thread_x-1];
		avg[2] = firesrc[(thread_y+1)*maxwidth + thread_x];
		if((thread_x+1) < 800)avg[3] = firesrc[(thread_y+1)*maxwidth + thread_x+1];

		avg[0] = (avg[1] + avg[2] + avg[3])/3;

		firedest[thread_y*maxwidth + thread_x] = avg[0];
	}

	pData[idx] = (((int)firedest[idx])&0xFF) << 16;

	return;
}
