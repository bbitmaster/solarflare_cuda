#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "kernels.h"
using namespace std;

__global__ void init_fire(hiprandState *state,float *firesrc,float *firedest);
__global__ void do_fire(unsigned int *pData,hiprandState *state,float *firesrc,float *firedest);

__global__ void setup_kernel(hiprandState *state);
hiprandState *devStates;

float *firebuff[2];
int flipstate;

int init_cuda(){

	hipMalloc((void **)&firebuff[0], SCREEN_HEIGHT * SCREEN_WIDTH *
	                  sizeof(float));

	hipMalloc((void **)&firebuff[1], SCREEN_HEIGHT * SCREEN_WIDTH *
	                  sizeof(float));

	//hipMemset(firebuff[0],0, SCREEN_HEIGHT * SCREEN_WIDTH *
	//                  sizeof(float));

	//hipMemset(firebuff[1],0, SCREEN_HEIGHT * SCREEN_WIDTH *
	//                  sizeof(float));

	hipMalloc((void **)&devStates, SCREEN_HEIGHT * SCREEN_WIDTH *
	                  sizeof(hiprandState));

	setup_kernel<<<SCREEN_HEIGHT,SCREEN_WIDTH>>>(devStates);

	init_fire<<<SCREEN_HEIGHT,SCREEN_WIDTH>>>(devStates,firebuff[0],firebuff[1]);

	flipstate=0;
	return 0;
}

int run_fire(unsigned int *pData){
	if(flipstate){
		do_fire<<<SCREEN_HEIGHT,SCREEN_WIDTH>>>(pData,devStates,firebuff[0],firebuff[1]);
		flipstate = 0;
	} else {
		do_fire<<<SCREEN_HEIGHT,SCREEN_WIDTH>>>(pData,devStates,firebuff[1],firebuff[0]);
		flipstate = 1;
	}
	return 0;
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void init_fire(hiprandState *state,float *firesrc,float *firedest){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState localState = state[id];
	unsigned int rand = hiprand(&localState);

    firesrc[id] = rand%128;
    firedest[id] = rand%128;
    state[id] = localState;
}

__global__ void do_fire(unsigned int *pData,hiprandState *state,float *firesrc,float *firedest){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_y = blockIdx.x;
	int thread_x = threadIdx.x;
	int maxwidth = blockDim.x;

	int maxheight = SCREEN_HEIGHT;

	hiprandState localState = state[idx];
	unsigned int rand = hiprand(&localState);
	state[idx] = localState;

	if(thread_y >= maxheight-1)
		firedest[idx] = rand&0xFF;
	__syncthreads();


	if(thread_y < maxheight-1){
		float avg[4];
		if((thread_x-1) >= 0)avg[1] = firesrc[(thread_y)*maxwidth + (thread_x-1)];
		avg[2] = firesrc[(thread_y+1)*maxwidth + thread_x];
		if((thread_x+1) < maxwidth)avg[3] = firesrc[(thread_y)*maxwidth + (thread_x+1)];

		avg[0] = (avg[1] + avg[2] + avg[3])/3;
		int rndcap = (avg[0]*0.035);//(4/138));
		rndcap += 1;

		if(avg[0] > 5.0)
			avg[0] += rand%rndcap;
		avg[0] -= 2.0;

		//avg[0] += rand%5;

		if(avg[0] > 255)avg[0] = 255;
		else if(avg[0] > 250)avg[0] = 0;

		if(avg[0] < 0)avg[0] = 255;
		firedest[thread_y*maxwidth + thread_x] = avg[0];
	}
	pData[idx] = ((int)firedest[idx]) << 16;

	return;
}
