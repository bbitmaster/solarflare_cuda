#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

__global__ void do_fire(unsigned int *pData,hiprandState *state,float *firebuff[2]);
__global__ void setup_kernel(hiprandState *state);
hiprandState *devStates;

float *firebuff[2];

int init_cuda(){

	hipMalloc((void **)&firebuff[1], 600 * 800 *
	                  sizeof(float));

	hipMalloc((void **)&firebuff[2], 600 * 800 *
	                  sizeof(float));

	hipMalloc((void **)&devStates, 600 * 800 *
	                  sizeof(hiprandState));
	setup_kernel<<<600,800>>>(devStates);

	return 0;
}

int run_fire(unsigned int *pData){
	do_fire<<<600,800>>>(pData,devStates,firebuff);
	return 0;
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void do_fire(unsigned int *pData,hiprandState *state,float *firebuff[2]){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_y = blockIdx.x;
	int thread_x = threadIdx.x;
	int maxwidth = blockDim.x;

	hiprandState localState = state[idx];
	//pData[idx] = ((blockIdx.x&0xff) << 8) + threadIdx.x;

	int rand = hiprand_uniform(&localState)*256;
	rand &= 0xFF;
	state[idx] = localState;

	if(thread_y >= 600-1)
		pData[idx] = (rand << 16);
	__syncthreads();

	if(thread_y < 600-1){
		pData[thread_y*maxwidth + thread_x] = pData[(thread_y+1)*maxwidth + thread_x];
	}

	return;
}
